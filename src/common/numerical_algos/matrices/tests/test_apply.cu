
#include <string>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <utils/init_cuda.h>
#include <communication/comm.h>
#include <communication/linear_partitioner.h>
#include <communication/parted_map.h>
#include <numerical_algos/vectors/block_vector.h>
#include <numerical_algos/matrices/brs_matrix.h>
#include <numerical_algos/matrices/brs_matrix_impl.h>

//TODO for_each default block size

typedef float                                               real;
static const t_tensor_field_storage                         storage = TFS_DEVICE;
typedef communication::linear_partitioner                   partitioner_t;
typedef communication::parted_map<partitioner_t>            map_t;
typedef numerical_algos::block_vector<real,storage,map_t>   vector_t;
typedef numerical_algos::brs_matrix<real,storage,map_t>     matrix_t;
typedef matrix_t::structure_type                            matrix_structure_t;

//TODO move it some common header and use in brs_matrix and brs_matrix_structure
int read_matrix_size(const std::string &fn)
{
    std::ifstream f(fn.c_str(), std::ifstream::in);
    if (!f) throw std::runtime_error("read_matrix_size: error while opening file " + fn);

    std::string     buf;
    int             algebraic_rows_n, algebraic_cols_n, 
                    algebraic_nonzeros_n;
    int             block_row_size_, block_col_size_;
    int             glob_rows_n_, glob_cols_n_;
    int             glob_nonzeros_n_;

    if (!getline(f,buf)) throw std::runtime_error("read_matrix_size: error while reading first line");
    if (!(f >> buf >> block_row_size_ >> block_col_size_)) throw std::runtime_error("read_matrix_size: error while read block sizes");
    if (block_row_size_ != block_col_size_) throw std::runtime_error("read_matrix_size: block is not square");
    if (!(f >> algebraic_rows_n >> algebraic_cols_n >> algebraic_nonzeros_n)) throw std::runtime_error("read_matrix_size: error while read sizes");
    if (algebraic_rows_n != algebraic_cols_n) throw std::runtime_error("read_matrix_size: matrix is not square");
    if (algebraic_rows_n%block_row_size_ != 0) throw std::runtime_error("read_matrix_size: matrix size is not divider of block size");
    if (algebraic_nonzeros_n%(block_row_size_*block_col_size_) != 0) throw std::runtime_error("read_matrix_size: matrix nonzero size is not divider of block size square");
    glob_rows_n_ = algebraic_rows_n/block_row_size_;
    glob_cols_n_ = algebraic_cols_n/block_col_size_;
    glob_nonzeros_n_ = algebraic_nonzeros_n/(block_row_size_*block_col_size_);

    return glob_rows_n_;
}

int main(int argc, char **args)
{
    if (MPI_Init(&argc, &args) != MPI_SUCCESS) {
        std::cout << "ERROR: MPI_Init call failed ; abort" << std::endl;
        return 1;
    }

    int comm_rank = communication::get_comm_rank(),
        comm_size = communication::get_comm_size();

    if (argc < 6) {
        if (comm_rank == 0)
            std::cout << "USAGE: " << std::string(args[0]) << " <matrix_fn> <vector_fn> <result_fn> <apply_type> <color_perm_fn>" << std::endl;
        if (MPI_Finalize() != MPI_SUCCESS) {
            std::cout << "WARNING: MPI_Finalize call failed" << std::endl;
            return 2;
        }
        return 0;
    }
    std::string             mat_fn(args[1]), vec_fn(args[2]), res_fn(args[3]);
    int                     apply_type = atoi(args[4]);
    std::string             color_perm_fn(args[5]);

    utils::init_cuda(1+comm_rank);
    hipsparseHandle_t        handle = 0;
    CUSPARSE_SAFE_CALL( hipsparseCreate(&handle) );

    int                     glob_size = read_matrix_size(mat_fn);
    partitioner_t           partitioner(glob_size, comm_size, comm_rank);
    map_t                   map(partitioner, true);

    vector_t                res, vec;
    matrix_structure_t      mat_str;
    matrix_t                mat;

    mat_str.pre_init_from_file(&map, mat_fn);
    map.complete();

    mat_str.init();
    mat_str.print_stat();
    mat.init(handle, &mat_str);
    mat.read_from_file(mat_fn);
    vec.init_from_file(map, vec_fn);
    res.init(map, vec.block_size());

    //vec.size()

    if (apply_type == 1)
        mat.apply(vec, res);
    else if (apply_type == 2)
        mat.apply_inverted_lower(vec, res);
    else if (apply_type == 3)
        mat.apply_inverted_upper(vec, res);
    else
        throw std::runtime_error("wrong apply_type argument");

    if (color_perm_fn != "none") mat_str.write_colored_perm(color_perm_fn);

    res.write_to_file(map, res_fn);

    if (MPI_Finalize() != MPI_SUCCESS) {
        std::cout << "WARNING: MPI_Finalize call failed" << std::endl;
        return 3;
    }

    return 0;
}